{%- macro memcpy_to_dev_args(p) -%}
  {{- p.devname() -}}, 
  {{- p.name(pre='h_') -}},
  {{- p.sizeof() -}},
  hipMemcpyHostToDevice
{%- endmacro -%}
{%- macro memcpy_from_dev_args(p) -%}
  {{- p.name(pre='h_') -}},
  {{- p.devname() -}}, 
  {{- p.sizeof() -}},
  hipMemcpyDeviceToHost
{%- endmacro -%}
#include "{{ name }}_cudawrapper.h"
#include "{{ name }}_cudaneighlist.h"
#include "{{ name }}_tpa.cu"
#include "{{ name }}_bpa.cu"
#include <cassert>
#include <cstdlib>
#include <cstdio>
#include <sstream>
#include <iostream>

#ifdef TRACE
#warning Turning TRACE on will affect timing results!
#include "cuPrintf.cu"
#endif

using namespace std;

{{ classname }}CudaWrapper::{{ classname }}CudaWrapper(
    int block_size,
    int N, int maxpage, int pgsize,
    {% for c in consts -%}
      {{ c.decl(pre='h_', include_dim=False) }},
    {% endfor -%}
    {% for p in params if p.is_type('P', 'RO') and not p.reload -%}
      {{ p.decl(pre='*h_', include_dim=False) }}{{ ',' if not loop.last }}
    {% endfor -%}
  ) :
  block_size(block_size),
  N(N),
  tpa_grid_size((N/block_size)+1),
  bpa_grid_size(N),
  // size of per-block array for
  {%- for p in params if p.is_type('P', 'SUM') -%}
    {{ ' ' }}{{ p.devname() }}{{ ',' if not loop.last }}
  {%- endfor %}
  bpa_shared_mem_size(
    {%- for p in params if p.is_type('P', 'SUM') -%}
      (block_size*{{ p.dim }}*sizeof({{ p.type }})){{ ' + ' if not loop.last }}
    {%- endfor -%}
  ),
  d_nl(new {{ classname }}CudaNeighList(block_size, N, maxpage, pgsize))
{
  {% for c in consts -%}
    hipMemcpyToSymbol("{{ c.devname() }}", &{{ c.name(pre='h_') }}, {{ c.sizeof() }}, 0, hipMemcpyHostToDevice);
  {% endfor %}
  {% for p in params if p.is_type('P', '-') -%}
    hipMalloc((void **)&{{ p.devname() }}, {{ p.sizeof() }});
  {% endfor %}
  {% for p in params if p.is_type('P', 'RO') and not p.reload -%}
    hipMemcpy({{ memcpy_to_dev_args(p) }});
  {% endfor %}
#if DEBUG
  cerr << "[DEBUG] Kernel TpA parameters grid_size=" << tpa_grid_size << " block_size=" << block_size << endl;
  cerr << "[DEBUG] Kernel BpA parameters grid_size=" << bpa_grid_size << " block_size=" << block_size << endl;
#endif
}

{{ classname }}CudaWrapper::~{{ classname }}CudaWrapper() {
  {% for p in params if not p.is_type('N', '-') -%}
    hipFree({{ p.devname() }});
  {% endfor %}
  delete(d_nl);
}

void {{ classname }}CudaWrapper::refill_neighlist(
  int *h_numneigh,
  int **h_firstneigh,
  int **h_pages,
  int maxpage,
  {% for p in params if p.is_type('N', '-') -%}
    {{ p.pages() }}{{ ',' if not loop.last }}
  {% endfor -%}
) {
  d_nl->reload(h_numneigh, h_firstneigh, h_pages, maxpage,
      {%- for p in params if p.is_type('N', '-') -%}
        {{ p.name(pre='h_',suf='pages') }}{{ ', ' if not loop.last }}
      {%- endfor -%}
  );
}

void {{ classname }}CudaWrapper::run(
  kernel_decomposition kernel,
  {% for p in params -%}
    {%- if p.is_type('P', 'RO') and p.reload -%}
      {{ p.decl(pre='*h_', include_dim=False) }}{{ ',' if not loop.last }}
    {%- elif p.is_type('P', 'RW') or p.is_type('P', 'SUM') -%}
      {{ p.decl(pre='*h_', include_dim=False) }}{{ ',' if not loop.last }}
    {%- elif p.is_type('N', 'RO') and p.reload -%}
      {{ p.pages() }}{{ ',' if not loop.last }}
    {%- elif p.is_type('N', 'RW') -%}
      {{ p.pages() }}{{ ',' if not loop.last }}
    {%- else -%}
      // {{ p.name() }} is not reloaded
    {%- endif %}
  {% endfor -%}
) {
  {% for p in params if p.is_type('P', '-') and p.reload -%}
    hipMemcpy({{ memcpy_to_dev_args(p) }});
  {% endfor -%}
  {% for p in params if p.is_type('P', 'RW') or p.is_type('P', 'SUM') -%}
    hipMemcpy({{ memcpy_to_dev_args(p) }});
  {% endfor %}

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Pre-compute-kernel error: %s.\n", hipGetErrorString(err));
    exit(1);
  }
#ifdef TRACE
  cudaPrintfInit();
#endif
  if (kernel == TPA) {
    {{ name }}_tpa<<<tpa_grid_size, block_size>>>(
      N,
      {% for p in params if p.is_type('P', 'RO') -%}
      {{ p.devname() }},
      {% endfor -%}
      d_nl->d_numneigh, d_nl->d_offset, d_nl->d_neighidx,
      {% for p in params if not p.is_type('P', 'RO') -%}
        {%- if p.is_type('N', '-') -%}
      d_nl->{{ p.devname() }}{{ ', ' if not loop.last }}
        {%- else -%}
      {{ p.devname() }}{{ ', ' if not loop.last }}
        {%- endif -%}
      {% endfor -%}
    );
  } else if (kernel == BPA) {
    {{ name }}_bpa<<<bpa_grid_size, block_size, bpa_shared_mem_size>>>(
      N,
      {% for p in params if p.is_type('P', 'RO') -%}
      {{ p.devname() }},
      {% endfor -%}
      d_nl->d_numneigh, d_nl->d_offset, d_nl->d_neighidx,
      {%- for p in params if not p.is_type('P', 'RO') %}
        {%- if p.is_type('N', '-') %}
      d_nl->{{ p.devname() }}{{ ', ' if not loop.last }}
        {%- else %}
      {{ p.devname() }}{{ ', ' if not loop.last }}
        {%- endif -%}
      {% endfor -%}
    );
  }
  hipDeviceSynchronize();
#ifdef TRACE
  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();
#endif
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Post-compute-kernel error: %s.\n", hipGetErrorString(err));
    exit(1);
  }

  {% for p in params if p.is_type('P', 'RW') or p.is_type('P', 'SUM') -%}
    hipMemcpy({{ memcpy_from_dev_args(p) }});
  {% endfor %}

  {% for p in params if p.is_type('N', 'RW') -%}
  if ({{ p.name(pre='h_',suf='pages') }} != NULL) {
    d_nl->unload_{{ p.name() }}({{ p.name(pre='h_',suf='pages') }});
  }
  {% endfor %}
}

