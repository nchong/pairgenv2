#include "hip/hip_runtime.h"
#include "cudaneighlist.h"
#include "thrust/scan.h"

#define PARANOID true
#if PARANOID
#include "scanref.h"
#include <cassert>
#endif

__global__ void decode_neighlist_p1(
  //inputs
  int nparticles,
  int **firstneigh, //nb: contains cpu pointers: do not dereference!
  int maxpage,
  int **pages,      //nb: contains cpu pointers: do not dereference!
  int pgsize,
  //outputs
  int *pageidx
) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid < nparticles) {
    int *myfirstneigh = firstneigh[tid];
    int mypage = -1;
    for (int p=0; p<maxpage; p++) {
      if ( (pages[p] <= myfirstneigh) &&
                       (myfirstneigh < (pages[p]+pgsize)) ) {
        mypage = p;
      }
    }
    pageidx[tid] = mypage;
  }
}

CudaNeighList::CudaNeighList(
  int block_size,
  int nparticles, int maxpage, int pgsize) :
  block_size(block_size),
  grid_size((nparticles/block_size)+1),
  nparticles(nparticles), maxpage(maxpage), pgsize(pgsize),

  d_numneigh_size(nparticles * sizeof(int)),
  d_firstneigh_size(nparticles * sizeof(int *)),
  d_pages_size(maxpage * sizeof(int *)),
  d_pagebreak_size(d_numneigh_size),
  d_pageidx_size(d_numneigh_size),
  d_offset_size(d_numneigh_size),
  d_neighidx_size(maxpage * pgsize * sizeof(int)),

  tload(0), tunload(0), tdecode(0)
{
  hipMalloc((void **)&d_numneigh, d_numneigh_size);
  hipMalloc((void **)&d_firstneigh, d_firstneigh_size);
  hipMalloc((void **)&d_pages, d_pages_size);
  hipMalloc((void **)&d_pageidx, d_pageidx_size);
  hipMalloc((void **)&d_offset, d_offset_size);
  hipMalloc((void **)&d_neighidx, d_neighidx_size);
}

CudaNeighList::~CudaNeighList() {
  hipFree(d_numneigh);
  hipFree(d_firstneigh);
  hipFree(d_pages);
  hipFree(d_pageidx);
  hipFree(d_offset);
  hipFree(d_neighidx);
}

/*
 * Make bigger device arrays
 * NB: does not update maxpage!
 */
void CudaNeighList::resize(int new_maxpage) {
  hipFree(d_pages);
  hipFree(d_neighidx);
  d_pages_size = new_maxpage * sizeof(int *);
  d_neighidx_size = new_maxpage * pgsize * sizeof(int);
  hipMalloc((void **)&d_pages, d_pages_size);
  hipMalloc((void **)&d_neighidx, d_neighidx_size);
}

/*
 *    CPU            DEV
 *
 *    numneigh ----> [d_numneigh] -------------------------------(scan) --> [d_offset]
 *                                                                 /
 *  .(if maxpage > 1)...........................                  /
 *  | firstneigh -->  d_firstneigh -- (decode) |-> [d_pageidx] --'
 *  `........                           /      |
 *          |                          /       |
 *    pages |------>  d_pages --------'        |
        \   |...................................
 *        \
 *         '-------> [d_neighidx]
 *
 */
void CudaNeighList::reload(int *numneigh, int **firstneigh, int **pages, int reload_maxpage) {
  // nb: we do not expect nparticles or pgsize to change
  // resize if necessary
  if (maxpage < reload_maxpage) {
    resize(reload_maxpage);
    maxpage = reload_maxpage;
  }

  hipMemcpy(d_numneigh, numneigh, d_numneigh_size, hipMemcpyHostToDevice);
  load_pages(d_neighidx, pages);
  thrust::device_ptr<int> thrust_numneigh(d_numneigh);
  thrust::device_ptr<int> thrust_offset(d_offset);

  if (maxpage == 1) {
    thrust::exclusive_scan(thrust_numneigh, thrust_numneigh + nparticles, thrust_offset);
    hipMemset(d_pageidx, 0, d_pages_size);
  } else {
    hipMemcpy(d_firstneigh, firstneigh, d_firstneigh_size, hipMemcpyHostToDevice);
    hipMemcpy(d_pages, pages, d_pages_size, hipMemcpyHostToDevice);
    decode_neighlist_p1<<<grid_size, block_size>>>(
      nparticles,
      d_firstneigh,
      maxpage,
      d_pages,
      pgsize,
      d_pageidx);
    thrust::device_ptr<int> thrust_pageidx(d_pageidx);
    thrust::exclusive_scan_by_key(
      thrust_pageidx,              // ] keys
      thrust_pageidx + nparticles, // ] 
      thrust_numneigh,             //vals
      thrust_offset);              //output
  }

#if PARANOID
  check_decode(numneigh, firstneigh, pages);
#endif
}

void CudaNeighList::check_decode(int *numneigh, int **firstneigh, int **pages) {
  if (maxpage == 1) {
    // simulate segmented scan
    int *expected_offset = new int[nparticles];
    exclusive_scan_host(expected_offset, numneigh, nparticles);
    // check
    int *offset = new int[nparticles];
    hipMemcpy(offset, d_offset, d_offset_size, hipMemcpyDeviceToHost);
    for (int i=0; i<nparticles; i++) {
      assert(offset[i] == expected_offset[i]);
    }

    // end-to-end check of decode
    int *neighidx = new int[maxpage*pgsize];
    hipMemcpy(neighidx, d_neighidx, d_neighidx_size, hipMemcpyDeviceToHost);
    for (int i=0; i<nparticles; i++) {
      for (int j=0; j<numneigh[i]; j++) {
        int expected = firstneigh[i][j];
        int myoffset = offset[i];
        int actual = neighidx[myoffset+j];
        assert(expected == actual);
      }
    }

    delete[] expected_offset;
    delete[] neighidx;
  } else {
    // simulate decode_neighlist_p1
    int *expected_pagebreak = new int[nparticles];
    int *expected_pageidx = new int[nparticles];
    for (int i=0; i<nparticles; i++) {
      int *myfirstneigh = firstneigh[i];
      int mypagebreak = 0;
      int mypage = -1;
      for (int p=0; p<maxpage; p++) {
        mypagebreak |= (myfirstneigh == pages[p] ? 1 : 0);
        if ( (pages[p] <= myfirstneigh) &&
                         (myfirstneigh < (pages[p]+pgsize)) ) {
          mypage = p;
        }
      }
      expected_pagebreak[i] = mypagebreak;
      expected_pageidx[i] = mypage;
    }
    // check
    int *pageidx = new int[nparticles];
    hipMemcpy(pageidx, d_pageidx, d_pageidx_size, hipMemcpyDeviceToHost);
    for (int i=0; i<nparticles; i++) {
      assert (pageidx[i] == expected_pageidx[i]);
    }

    // simulate segmented scan
    int *expected_offset = new int[nparticles];
    segmented_exclusive_scan_host(expected_offset, numneigh, expected_pagebreak, nparticles);
    // check
    int *offset = new int[nparticles];
    hipMemcpy(offset, d_offset, d_offset_size, hipMemcpyDeviceToHost);
    for (int i=0; i<nparticles; i++) {
      assert(offset[i] == expected_offset[i]);
    }

    // end-to-end check of decode
    int *neighidx = new int[maxpage*pgsize];
    hipMemcpy(neighidx, d_neighidx, d_neighidx_size, hipMemcpyDeviceToHost);
    for (int i=0; i<nparticles; i++) {
      for (int j=0; j<numneigh[i]; j++) {
        int expected = firstneigh[i][j];
        int mypage = pageidx[i];
        int myoffset = offset[i];
        int actual = neighidx[(mypage*pgsize)+myoffset+j];
        assert(expected == actual);
      }
    }

    delete[] expected_pagebreak;
    delete[] expected_pageidx;
    delete[] pageidx;
    delete[] expected_offset;
    delete[] offset;
    delete[] neighidx;
  }
}
